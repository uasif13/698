#include "hip/hip_runtime.h"
/*
  Andrew Sohn
  10/27/2025
  CS698 MPI+CUDA Programming
  *
  The MPI+CUDA program compiles and passes the test because they are all zeros.
  Fill the functions.

  NOTE:
  need to place nvidia Common directory two dirs above the current dir
  or 
  change the Makefile reference of Common
*/
#include <iostream>
using std::cerr;
using std::endl;

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <unistd.h>
#include <sys/time.h>

#define TILE_WIDTH 4
#define MAX_TILE_WIDTH 16
#define THREADS_PER_BLOCK 256

#define MAX_BUF_SIZE 1<<25
extern "C" {
  int matrix_multiply_cuda(int nprocs, int my_rank,int n, int my_work,int *h_A,int *h_B,int *h_C,int gx_dim,int gy_dim,int bx_dim,int by_dim );
}
void output_vector(int* data, int datasize);
int matrix_multiply_cpu(int my_rank,int *a, int *b, int *c, int n, int my_work) {
    int i, j, k, sum = 0;
    for (i = 0; i < my_work*n; i ++){
    	sum = 0;
	j = (i/n)*n;
	k = i%n;

	while (j < (i/n)*n+n && k < n*n){
	      sum += a[j]*b[k];
	      j ++;
	      k = k + n;
	}
	c[i] = sum;
    }

  return 0;
}

int compare_cpu(int my_rank, int *host, int *dev, int n, int my_work) {
  int i,j,idx;

  for (i=0; i<my_work; i++) {
    for (j=0; j<n; j++) {
      idx = i*my_work + j;
      if (dev[idx] != host[idx]) {
	printf("DIFFERENT: rank=%d: dev[%d][%d]=%d != host[%d][%d]=%d\n", \
	       my_rank,i,j,dev[idx],i,j,host[idx]);
	return 0;
      }
    }
  }

  return 1;
}

__global__ void mat_mult_cuda(int my_rank, int a_width,int my_work, int *d_a, int *d_b, int *d_c, int tile_width){
  /* 
  __shared__ int a_shared[][] ...
  __shared__ int b_shared[][] ...
 */
	extern	__shared__ int a_shared[];
	extern __shared__ int b_shared[];
	int shift = my_rank*my_work;
	int n_sq = a_width*a_width;
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	int j = blockIdx.y*blockDim.y+threadIdx.y;
			
	int index = i+j*a_width;

	// does not include zeros
	if (shift/a_width+j < a_width)
	   for (int n = j*a_width ; n < (j+1)*a_width; n++){
		a_shared[n] = d_a[n];
		}
	    	 
	


	// does not include zeros
	for (int k = i; k < n_sq; k=k+a_width)
	    b_shared[my_work+k] = d_b[k];

	printf("%d %d: %d\n", i, j, index);	
	if (i == 0 && j == 0){
	   printf("length of a: %d\n",my_work);
	   for (int m = 0; m < a_width; m++)
	          printf("a_shared: %d ",a_shared[m]);
	   }

	
	__syncthreads();
	if (i == 0 && j == 0){
	   for (int m = 0; m < a_width * a_width; m=m+a_width)
	       printf("b_shared: %d ",b_shared[m]);
	   }

	__syncthreads();
	int sum = 0;
	for (int l = 0; l < a_width; l++){
	    int a_index = index/a_width*a_width+l;
	    int b_index = index%a_width+l*a_width;
	    if (shift+a_index < n_sq && b_index < n_sq)
	       sum = sum + a_shared[a_index]*b_shared[my_work+b_index];
	    if (i == 0 && j == 0) {
	        printf("%d %d %d: %d\n", index, a_index, b_index, sum);
	    }

	     
	  
	}
	d_c[index] = sum;
	}

void print_lst_cpu(int name,int rank,int n, int *l){
  int i=0;
  printf("CPU rank=%d: %d: ",rank,name);
  for (i=0; i<n; i++) printf("%x ",l[i]);
  printf("\n");
}

int matrix_multiply_cuda(int nprocs, int my_rank,int n, int my_work,int *h_A,int *h_B,int *h_C,int gx_dim,int gy_dim,int bx_dim,int by_dim ) {
  int cuda_prod=0;
  int *d_A, *d_B, *d_C;
  struct timeval timecheck;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop,0);
  if (my_rank == 0) {
  printf("\n**** properties: rank=%d *****\n",my_rank);
  printf("prop.name=%s\n", prop.name);
  printf("prop.multiProcessorCount=%d\n", prop.multiProcessorCount);
  printf("prop.major=%d minor=%d\n", prop.major, prop.minor);
  printf("prop.maxThreadsPerBlock=%d\n", prop.maxThreadsPerBlock);
  printf("maxThreadsDim.x=%d maxThreadsDim.y=%d maxThreadsDim.z=%d\n", prop.maxThreadsDim[0],prop.maxThreadsDim[1],prop.maxThreadsDim[2]);
  printf("prop.maxGridSize.x=%d maxGridSize.y=%d maxGridSize.z=%d\n", prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);
  printf("prop.maxThreadsPerMultiProcessor=%d\n", prop.maxThreadsPerMultiProcessor);
  printf("prop.totalGlobalMem=%u\n", prop.totalGlobalMem);
  printf("prop.regsPerBlock=%d\n", prop.regsPerBlock);
  printf("**** properties: rank=%d *****\n",my_rank);
  printf("\n");
  }


  unsigned int my_work_size = sizeof(int) * my_work;
  unsigned int mat_size = sizeof(int) * my_work*nprocs;
  printf("rank=%d: my_work=%d data_size=%d bytes\n",my_rank,my_work,my_work_size);

  long dev_start, dev_end, dev_elapsed;
  gettimeofday(&timecheck, NULL);
  dev_start = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec/ 1000;

  int *h_C_on_cpu = (int *) malloc(my_work_size);

  hipMalloc(reinterpret_cast<void **>(&d_A), my_work_size);
  hipMalloc(reinterpret_cast<void **>(&d_B), mat_size);
  hipMalloc(reinterpret_cast<void **>(&d_C), my_work_size);

  hipMemcpy(d_A, h_A, my_work_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, mat_size, hipMemcpyHostToDevice);

  by_dim = bx_dim;
  gx_dim = n/bx_dim;
  if (n%bx_dim != 0)
     gx_dim ++;
  gy_dim = n/(bx_dim*nprocs);
  if (n%(bx_dim*nprocs)!= 0)
     gy_dim ++;

  printf("bx_dim:%d by_dim:%d gx_dim:%d gy_dim:%d \n",bx_dim, by_dim,gx_dim,gy_dim);
  dim3 grid(gx_dim,gy_dim);
  dim3 threads(bx_dim,by_dim);

  mat_mult_cuda<<<grid,threads,my_work_size>>>(my_rank,n,my_work,d_A, d_B, d_C,by_dim);

  hipMemcpy(h_C,d_C,my_work_size, hipMemcpyDeviceToHost);

  gettimeofday(&timecheck, NULL);
  dev_end = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
  dev_elapsed = dev_end - dev_start;
  
  printf("dev time: rank=%d: %d procs: %ld msecs\n",
	   my_rank, nprocs, dev_elapsed);

  fflush(stdout);
    printf("vector D with size %d\n", n*my_work);
//  output_vector(h_A,n*my_work);
  printf("vector C with size %d\n", n*my_work);
  //output_vector(h_C,n*my_work);
  matrix_multiply_cpu(my_rank,h_A,h_B,h_C_on_cpu,n,my_work);
  printf("vector C_on_cpu with size %d\n", n*my_work);
 // output_vector(h_C_on_cpu,n*my_work);
  if (compare_cpu(my_rank,h_C_on_cpu,h_C,n,my_work)) /* h_C is from dev */
    printf("\nrank=%d: Test CPU: PASS: host == dev\n", my_rank);
  else
    printf("\nrank=%d: Test CPU: FAIL: host != dev\n", my_rank);

  fflush(stdout);

  return cuda_prod;

}

void output_vector(int* data, int datasize){
     for (int i = 0; i < datasize; i++){
     	 printf("%d ",data[i]);
     }
     printf("\n");
}

void init_vec(int* data, int datasize) {
     for (int i = 0; i < datasize; i++){
     	 data[i] = rand() & 0xF;
     }
}
/*
int main(void){
    int n = 4;
    int numprocs = 2;
    int my_work = n * n / numprocs;
    int * a = new int [my_work];
    int * c = new int [my_work];
    int * d = new int [my_work];
    int * e = new int [my_work];
    init_vec(a,my_work);
    init_vec(c,my_work);
    init_vec(d,my_work);
    init_vec(e,my_work);
    int * f = new int [my_work];
    int * g = new int [my_work];
    int * h = new int [my_work];
    int * i = new int [my_work];



    int * b = new int [n*n];
    init_vec(b,n*n);
    output_vector(b,n*n);


    int tile_width = n/numprocs;
    int bx_dim = tile_width;
    int by_dim = tile_width;
    int gx_dim = n/bx_dim;
    int gy_dim = n/(bx_dim*numprocs);

    
    matrix_multiply_cuda(2,0, n,my_work, a, b, f, bx_dim, by_dim, gx_dim,gy_dim);
    matrix_multiply_cuda(2,1, n,my_work, c, b, g, bx_dim, by_dim, gx_dim,gy_dim);
//    matrix_multiply_cuda(4,3, n,my_work, d, b, h, bx_dim, by_dim, gx_dim,gy_dim);
  //  matrix_multiply_cuda(4,4, n,my_work, e, b, i, bx_dim, by_dim, gx_dim,gy_dim);
    output_vector(f,my_work);
    output_vector(g,my_work);
    output_vector(h,my_work);
    output_vector(i,my_work);
}*/
/*************************************************
  End of file
*************************************************/
